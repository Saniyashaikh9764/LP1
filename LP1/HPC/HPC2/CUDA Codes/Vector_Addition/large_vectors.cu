//CUDA by Example Page 63


#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void addition(int *a, int *b, int *c, int n) {
	int large_id = blockIdx.x * blockDim.x + threadIdx.x;
	while (large_id < n) {
	//if(large_id < n) {
		c[large_id] = a[large_id] + b[large_id];
		large_id += blockDim.x*gridDim.x;
	}
}

int main(void) {
	int n;
	cin>>n;
	//int a[n],b[n],c[n];
	int *a, *b, *c;
	a = (int *)malloc(n * sizeof(int));
	b = (int *)malloc(n * sizeof(int));
	c = (int *)malloc(n * sizeof(int));
	for(int i = 0; i < n; i++) {
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}
	int *dev_a, *dev_b, *dev_c;
	hipMalloc(&dev_a, n * sizeof(int));
	hipMalloc(&dev_b, n * sizeof(int));
	hipMalloc(&dev_c, n * sizeof(int));
	hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);
	//cudaMemcpy(dev_c, c, n * sizeof(int), cudaMemcpyHostToDevice);
	addition<<<128,128>>>(dev_a, dev_b, dev_c, n);
	hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i = 0; i < n; i++) {
		cout<<a[i]<<"+"<<b[i]<<"="<<c[i]<<endl;
	}
	
	//verify that gpu did work
	int count = 0;
	bool success = true;
	for(int i = 0; i < n; i++) {
		if((a[i] + b[i]) != c[i]) {
			cout<<"Error in "<<a[i]<<"+"<<b[i]<<"="<<c[i]<<endl;
			success = false;
			count++;
		}
	}
	if (success) cout<<"We did it"<<endl;
	cout<<"Number of errors: "<<count<<endl;
	
	
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}

//nvcc large_vectors.cu
//./a.exe
//nvprof ./a.exe