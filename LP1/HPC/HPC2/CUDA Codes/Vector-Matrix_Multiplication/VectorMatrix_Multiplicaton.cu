
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void vecMat(int *a, int *b, int *c, int n) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int sum = 0;
	for (int j = 0; j < n; j++) {
		sum += a[row * n + j] * b[j];
	}
	c[row] = sum;
}

int main() {
	int n;
	cin >> n;
	int *a = new int[n * n];
	int *b = new int[n];
	int *c = new int[n];
	int size = n * sizeof(int);
	cout<<"Matrix A: "<<endl;
	for (int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
			cin >> a[i * n + j];
		}
	}

	cout<<"Matrix A is: "<<endl;
	for(int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
			cout << "a[" << i * n + j << "] = " << a[i * n + j] << " ";
		}
		cout << endl;
	}

	cout<<"Vector B: "<<endl;
	for(int i = 0; i < n; i++) {
		cin >> b[i];
	}

	cout<<"Vector B is: "<<endl;
	for(int i = 0; i < n; i++) {
		cout << "b[" << i << "] = " <<b[i] << " ";
	}
	cout<<endl;
	
	int *dev_a, *dev_b, *dev_c;
	hipMalloc(&dev_a, n * size);
	hipMalloc(&dev_b, size);
	hipMalloc(&dev_c, size);

	hipMemcpy(dev_a, a, n * size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	dim3 grid_dim(n, n, 1);
	vecMat <<< grid_dim, 1 >>> (dev_a, dev_b, dev_c, n);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	cout << "Output: " << endl;
	for(int i = 0; i < n; i++) {
		cout<< "c[" << i << "] = " << c[i] <<" ";
	}
}
