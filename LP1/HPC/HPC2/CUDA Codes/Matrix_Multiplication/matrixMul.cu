
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void matMul(int *a, int *b, int *c, int n) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int sum = 0;
    for(int j = 0; j < n; j++) {
        sum += a[row * n + j] * b[j * n + col];
    }
    c[n*row + col] = sum;
}

int main() {
    int n;
    cin>>n;
    int *a = new int[n * n];
    int *b = new int[n * n];
    int *c = new int[n * n];
    int size = n * n * sizeof(int);
    cout<<"Matrix A: "<<endl;
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            cin >> a[i * n + j];
        }
    }
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            cout<<"a["<<i * n + j<<"] = "<< a[i * n + j]<<" ";
        }
        cout<<endl;
    }
    cout<<"Matrix B: "<<endl;
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            cin >> b[i * n + j];
        }
    }
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            cout<<"b["<<i * n + j<<"] = "<< b[i * n + j]<<" ";
        }
        cout<<endl;
    }
    int *dev_a, *dev_b, *dev_c;
    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&dev_c, size);
    
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    dim3 grid_dim(n, n, 1);
    matMul<<<grid_dim, 1>>> (dev_a, dev_b, dev_c, n);
    
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    cout<<"Output: "<<endl;
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            cout<< c[i * n + j]<<" ";
        }
        cout<<endl;
    }
    
}