
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;


//standard deviation = sqrt(summation(x-mean^2) / n)
//Did not figure out why this code doesn't work with threads
__global__ void standard_deviation(int *a, float *b, float mean, int n) {
    int tid = blockIdx.x;
    //int tid - threadIdx.x;
    b[0] = 0.0;
    for(int i = tid; i < n; i++) {  
        b[0] += (a[i] - mean) * (a[i] - mean);
        //printf("b[%d] = %d, a[%d] = %d", i, b[0], i, a[i]);
    }
    b[0] = b[0]/n;
}

int main() {
    int n;
    cin>>n;
    //int a[n]; //does not work on some cuda versions
    int *a = (int *)malloc(n * sizeof(int));
    cout<<"The input numbers are: "<<endl;
    for(int i = 0; i < n; i++) {
        a[i] = i+1;
        cout<<a[i]<<"\t";
    }
    cout<<endl;
    float mean = (n + 1)/2;
    cout<<"Mean: "<<mean<<endl;
    int *dev_a;
    float *dev_b;
    hipMalloc(&dev_a, n * sizeof(int));
    hipMalloc(&dev_b, sizeof(float));

    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    standard_deviation<<<n, 1>>>(dev_a, dev_b, mean, n);
    float *ans = (float *)malloc(sizeof(float));
    hipMemcpy(ans, dev_b, sizeof(float), hipMemcpyDeviceToHost);
    cout<<"The answer is: "<< sqrt(ans[0])<<endl;
}