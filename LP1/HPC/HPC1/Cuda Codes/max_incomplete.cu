#include "hip/hip_runtime.h"
#include<iostream>
#include<chrono>
#include<limits>
using namespace std;
using namespace std::chrono;

__global__ void find_maximum(float *arr, float *max, int *mutex, unsigned int n) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x;
    unsigned int offset = 0;

    __shared__ float cache[256];
    float temp = 100000000000.99;
    while (index + offset < n) {
        temp = fmaxf(temp, arr[index + offset]);
        offset += stride;
    }
    cache[threadIdx.x] = temp;
    __syncthreads();
    //Reduction
    unsigned int i = blockDim.x / 2;
    while(i != 0) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x]);
        }
        __syncthreads();
        i /= 2;
    }
    if(threadIdx.x == 0) {
        while(atomicCAS(mutex, 0, 1)!= 0); //lock
        *max = fmaxf(*max, cache[0]);
        atomicExch(mutex, 0);
    }
}

void find_maximum_CPU(float *a, int n) {
    float max = FLT_MAX;
    for(int i = 0; i < n; i++) {
        if(a[i] < max) {
            max = a[i];
        }
    }
    cout<<"\nThe max number (CPU) is: "<<max<<endl;
}

int main() {
    float *a, *dev_a, *max, *dev_max;
    int *dev_mutex;
    int n = 1024 * 1024 * 20;
    a = (float *)malloc(n * sizeof(float));
    max = (float *)malloc(sizeof(float));
    for(int i = 0; i < n; i++) {
        a[i] = float(rand()) + 69.0f;
    }
    //Max with CPU
    auto startCPU = high_resolution_clock::now();
    find_maximum_CPU(a, n);
    auto stopCPU = high_resolution_clock::now();
    cout<<"\nTime elapsed on CPU: "<<duration_cast<microseconds>(stopCPU - startCPU).count()<<endl;
    
    hipMalloc(&dev_a, n * sizeof(float));
    hipMalloc(&dev_max, sizeof(float));
    hipMalloc(&dev_mutex, sizeof(int));
    hipMemset(dev_max, 0, sizeof(float));
    hipMemset(dev_mutex, 0, sizeof(int));
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridSize = 256;
    dim3 blockSize = 256;
    auto startGPU = high_resolution_clock::now();
    find_maximum<<<gridSize, blockSize>>> (dev_a, dev_max, dev_mutex, n);
    auto stopGPU = high_resolution_clock::now();
    hipMemcpy(max, dev_max, sizeof(float), hipMemcpyDeviceToHost);
    cout<<"\nThe Max number (GPU): "<<*max<<endl;
    cout<<"\nTime elapsed on GPU: "<<duration_cast<microseconds>(stopGPU - startGPU).count()<<endl;
}