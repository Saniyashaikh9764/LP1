
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>

using namespace std;

__global__ void sum(int *a, int *b, int n)
{
	int tid = threadIdx.x;
	int sum = 0;

    for(int i=tid; i<min(tid+256,n); i++)
    //tid starts at 0, from i = 0 till whatever is lower, n or 256, keep going to next thread till thread = 256
    //since 256 number of threads is max number of threads one block can have
	{
		sum += a[i];
	}
    b[tid]=sum; //cannot have b[0]=b[0] + a[i] instead of sum += a[i]
                //because it is adding the last element of array twice to the "sum" variable
}

int main()
{
	cout<<"Enter the no of elements"<<endl;
	int n;
    cin>>n;

    //int a[n];
    //cannot create an array as above in nvcc 10.1, so we have created *a 
    int *a;
    a = (int *)malloc(n * sizeof(int));

    //initializing values in the host array, feel free to input what you want
	for(int i=0; i<n; i++)
	{
        a[i] = i;
        //a[i] = rand();
    }
    
	int *dev_a,*dev_b;
    int size = n * sizeof(int);
    
    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, sizeof(int));
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    
    sum<<<1,n>>>(dev_a, dev_b, n);
    
	int *add;
    add = (int *)malloc(sizeof(int));
	hipMemcpy(add, dev_b, sizeof(int), hipMemcpyDeviceToHost);
    cout<<"The sum is  "<<add[0]<<endl;

    float mean = 0;
    //we want mean to be float, so we need RHS to be float
	mean = add[0]/(n*1.0);
	cout<<"The mean is   "<<mean<<endl;
}
