
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void minimum(int *a, int *b, int n) {
    int tid = threadIdx.x;
 //   int min_limit = 99999;
	int max=0;

    for(int i = tid; i < min(tid + 256, n); i++) {
    //for(int i = 0; i < n; i++) will work just fine
    
  /*  if (min_limit > a[i]) {					//condition for minimum 
            min_limit = a[i];
        }*/
        //printf("Min limit in %d = %d\n", i,min_limit); //debugging purpose
	if(max<a[i])
		{
		max=a[i];
		}
    }
   // b[tid] = min_limit;
	b[tid] = max;
}

int main() {
    cout << "Enter the size of the array" << endl;
    int n;	
    cin >> n;

	hipEvent_t start,end;

    //int a[n]; //does not work in some cuda versions
    int *a = (int *)malloc(n * sizeof(int));
    for(int i = 0; i < n; i++) {
        a[i] = i;
       //a[i] = rand();
    }
    //checking the values of a[i] to see what has been given in the input
    cout<<"The input values given are:"<<endl;
    for(int i = 0; i < n; i++) {
        cout<<a[i]<<"\t";
    }
    cout<<endl;
    int *dev_a, *dev_b;
    int size = n * sizeof(int);

    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, sizeof(int));

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);
    minimum<<<1, n>>>(dev_a, dev_b, n);
	hipEventRecord(end);
	hipEventSynchronize(end);
	float time=0;
	hipEventElapsedTime(&time,start,end);
	cout<<"Time taken is"<<time<<"\n";
    int *ans = (int *)malloc(sizeof(int));
    hipMemcpy(ans, dev_b, sizeof(int), hipMemcpyDeviceToHost);
    cout<<"The minimum element is: "<<ans[0]<<endl;
}
