#include "hip/hip_runtime.h"
#include<iostream>
#include<chrono>
using namespace std;
using namespace std::chrono; //for execution time checking

__global__ void sumGPU(int *a, long int *b, int n) {
    for(int i = 0; i < n; i++) {
        b[0] += a[i];
    }
    /*int tid = threadIdx.x;
    while (tid < n) {
        b[0] += a[tid];
    }*/
}

void sumCPU(int *a, int n) {
    long int b = 0;
    for(int i = 0; i < n; i++) {
        b += a[i];
    }
    cout<<"CPU sum= "<<b<<endl;
}

int main() {
    int *a, *dev_a;
    long int *dev_b;
    int n;
    cin >> n;
    //allocate a with memory of size of n integers
    a = (int *)malloc(n * sizeof(int)); 
    for(int i = 0; i < n; i++) {
        //a[i] = i;
        a[i] = rand;
    }
    //Allocate memory in CUDA to device dev_a
    hipMalloc(&dev_a, n * sizeof(int));
    //Allocate memory in CUDA to device dev_b
    hipMalloc(&dev_b, sizeof(long int));
    //Copy data from host a to device dev_a
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);

    //Compute sum of array on GPU
    auto startGPU = high_resolution_clock::now();
    sumGPU<<<1, n>>>(dev_a, dev_b, n);
    auto stopGPU = high_resolution_clock::now();

    //Print sum of GPU
    long int *output;
    output = (long int *)malloc(sizeof(long int));
    hipMemcpy(output, dev_b, sizeof(long int), hipMemcpyDeviceToHost);
    cout<<"GPU sum = "<<output[0]<<endl;
    cout<<"Time required by GPU: "<<duration_cast<microseconds>(stopGPU - startGPU).count() << endl;

    //Compute sum of array on CPU
    auto startCPU = high_resolution_clock::now();
    sumCPU(a, n);
    auto stopCPU = high_resolution_clock::now();
    cout<<"Time required by CPU: "<<duration_cast<microseconds>(stopGPU - startGPU).count() << endl;

}